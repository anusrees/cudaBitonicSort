#include <hip/hip_runtime.h>
#include <bits/stdc++.h>

#define BLOCKS 32768*2
#define THREADS 256
#define SIZE BLOCKS*THREADS

using namespace std;

__host__ void printArr(int *arr, int size);
__host__ void randomArrGenerator(int *arr, int size);
__host__ void checkSorted(int *arr, int size);

__device__ void swapCu(int &a, int &b)
{
	int temp = a;
	a = b;
	b = temp;
}

//bitonic sort on GPU
__global__ void bitonicSortCu(int *arr, int i, int j, int size)
{
	int k = threadIdx.x + blockIdx.x*blockDim.x;

	if(k<size && k%(j<<1) < j)
	{
		bool descending = (k/i)%2;

		if(descending && arr[k] < arr[k+j])
			swapCu(arr[k], arr[k+j]);
		else if(!descending && arr[k] > arr[k+j])
			swapCu(arr[k], arr[k+j]);
	}
}

void bitonicSortParallel(int *arr, int size)
{
	for(int i=2; i<=size; i*=2)
		for(int j=i/2; j>=1; j/=2)
			bitonicSortCu<<<BLOCKS, THREADS>>>(arr, i, j, size);
}

int main(int argc, char const *argv[])
{
	int *d_arr;
	int *arr = new int[SIZE];

	randomArrGenerator(arr, SIZE);

	hipMalloc(&d_arr, sizeof(int)*SIZE);
	hipDeviceSynchronize();

//start timer here
	hipMemcpyAsync(d_arr, arr, sizeof(int)*SIZE, hipMemcpyHostToDevice);
	bitonicSortParallel(d_arr, SIZE);
	hipMemcpyAsync(arr, d_arr, sizeof(int)*SIZE, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
//end timer here

	checkSorted(arr, SIZE);

	return 0;
}

//Auxilliary CPU functions
__host__ void checkSorted(int *arr, int size)
{
	for(int i=1; i<size; i++)
		if(arr[i] < arr[i-1])
		{
			cout << "sorting unsuccessful\n";
			return;
		}

	cout << "sorting successful\n";
}

__host__ void randomArrGenerator(int *arr, int size)
{
	for(int i=0; i<size; i++)
		arr[i] = rand()%1000;
}

__host__ void printArr(int *arr, int size)
{
	for(int i=0; i<size; i++)
		cout << arr[i] << " ";
	cout << endl;
}

__host__ void swap(int &a, int &b)
{
	int temp = a;
	a = b;
	b = temp;
}

//bitonic sort on CPU
__host__ void bitonicSort(int *arr, int size)
{
	if(size > 1)
	{
		for(int i=2; i<=size; i*=2)
		{
			for(int j=i/2; j>=1; j/=2)
			{
				for(int k=0; k<size; k++)
				{
					if(k%(j<<1) < j)
					{
						bool descending = (k/i)%2;

						if(descending && arr[k] < arr[k+j])
							swap(arr[k], arr[k+j]);
						else if(!descending && arr[k] > arr[k+j])
							swap(arr[k], arr[k+j]);
					}
				}
			}
		}
	}
}
